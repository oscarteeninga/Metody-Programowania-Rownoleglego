#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N 10

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void **) &dev_a, N * sizeof(int));
    hipMalloc((void **) &dev_b, N * sizeof(int));
    hipMalloc((void **) &dev_c, N * sizeof(int));
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 2;
    }
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice);
    add <<<1, N>>>(dev_a, dev_b, dev_c);
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        printf("%d+%d=%d\n", a[i], b[i], c[i]);
    }
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}