#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <helper_timer.h>
#include <iostream>
#include <chrono>

using namespace std;


__global__ void add(int *a, int *b, int *c, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}



int** cuda(int n, int gridSize, int blockSize){

    StopWatchInterface *timer=NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    int * a = new int[n];
    int * b = new int[n];
    int * c = new int[n];
    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void **) &dev_a, n * sizeof(int));
    hipMalloc((void **) &dev_b, n * sizeof(int));
    hipMalloc((void **) &dev_c, n * sizeof(int));
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i * 2;
    }
    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, n * sizeof(int), hipMemcpyHostToDevice);
    add <<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, n);
    hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    float time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    printf ("Time for the kernel: %f ms\n", time);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    int ** result = new int*[3];
    result[0] = a;
    result[1] = b;
    result[2] = c;
    return result;
}

int **cpu(int n){
    int * a = new int[n];
    int * b = new int[n];
    int * c = new int[n];

    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i * 2;
        c[i] = 0;
    }

    auto start_time = chrono::high_resolution_clock::now();

    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }

    auto end_time = chrono::high_resolution_clock::now();
    auto time = end_time - start_time;
    cout << "gpu\t" << n << "\t" << time << endl;

    int ** result = new int*[3];
    result[0] = a;
    result[1] = b;
    result[2] = c;
    return result;
}

int checkResults(int** cuda, int** cpp, int n){

    for(int i = 0; i < 3; ++i){
        for (int j = 0; j < n; ++j){
            if(cpp[i][j] != cuda[i][j]){
                return 0;
            }
        }
    }
    return 1;
}

int main(int argc, char* argv[]) {

    if (argc < 4) {
        cout << "Proper format is: <n> <grid_size> <block_size>" << endl;
        return 1;
    }

    int n = atoi(argv[1]);
    int gridSize = atoi(argv[2]);
    int blockSize = atoi(argv[3]);

    int** cudaResult = cuda(n, gridSize, blockSize);
    int** cpuResult = cpu(n);

    if(checkResults(cudaResult, cpuResult, n)) {
        cout << "Results are equal." << endl;
    } else {
        cout << "Results are NOT equal!" << endl;
    }
    return 0;
}